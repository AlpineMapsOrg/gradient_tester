#include "hip/hip_runtime.h"
/****************************************************************************
 *  Copyright (C) 2023 Adam Celarek (github.com/adam-ce, github.com/cg-tuwien)
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy
 *  of this software and associated documentation files (the "Software"), to deal
 *  in the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 *  of the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in
 *  all copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *  THE SOFTWARE.
 ****************************************************************************/

#include <stroke/unittest/gradcheck.h>

#include <catch2/catch_test_macros.hpp>
#include <stroke/gaussian.h>
#include <stroke/grad/gaussian.h>
#include <stroke/linalg.h>
#include <stroke/unittest/random_entity.h>

using Scalar = double;
using Vec2 = glm::vec<2, Scalar>;
using Vec3 = glm::vec<3, Scalar>;

namespace alpine {
struct GeomData {
    Vec2 p0;
    Vec2 p1;
    Vec2 p2;

    bool is_polygon;
};

Scalar sdf(const GeomData& data, const Vec2& uv)
{
    Vec2 e0 = data.p1 - data.p0;
    Vec2 v0 = uv - data.p0;
    Vec2 pq0 = v0 - e0 * glm::clamp(glm::dot(v0, e0) / glm::dot(e0, e0), Scalar(0), Scalar(1));

    Scalar poly_sign = 1.0;
    Scalar result = 1.0;

    if (data.is_polygon) {
        Vec2 e1 = data.p2 - data.p1;
        Vec2 e2 = data.p0 - data.p2;
        Vec2 v1 = uv - data.p1;
        Vec2 v2 = uv - data.p2;
        Vec2 pq1 = v1 - e1 * glm::clamp(glm::dot(v1, e1) / glm::dot(e1, e1), Scalar(0), Scalar(1));
        Vec2 pq2 = v2 - e2 * glm::clamp(glm::dot(v2, e2) / glm::dot(e2, e2), Scalar(0), Scalar(1));
        Scalar s = glm::sign(e0.x * e2.y - e0.y * e2.x);
        Vec2 d0 = Vec2(glm::dot(pq0, pq0), s * (v0.x * e0.y - v0.y * e0.x));
        Vec2 d1 = Vec2(glm::dot(pq1, pq1), s * (v1.x * e1.y - v1.y * e1.x));
        Vec2 d2 = Vec2(glm::dot(pq2, pq2), s * (v2.x * e2.y - v2.y * e2.x));
        Vec2 d = min(min(d0, d1), d2);

        poly_sign = -glm::sign(d.y);
        result = d.x;
    } else {
        result = dot(pq0, pq0);
    }

    return sqrt(result) * poly_sign;
};

Vec3 sdf_with_grad(const GeomData& data, const Vec2& uv, Scalar incoming_grad)
{
    const auto grad_self_dot = [](const Vec2& v, Scalar incoming_grad) {
        return Scalar(2) * v * incoming_grad;
    };

    Vec2 e0 = data.p1 - data.p0;
    Vec2 v0 = uv - data.p0;
    const auto dot0 = glm::dot(v0, e0);
    const auto one_over_dot0 = 1 / glm::dot(e0, e0);
    const auto div0 = dot0 * one_over_dot0;
    Vec2 pq0 = v0 - e0 * glm::clamp(div0, Scalar(0), Scalar(1));

    Scalar poly_sign = 1.0;
    Scalar distance_sq = 1.0;

    if (data.is_polygon) {
        Vec2 e1 = data.p2 - data.p1;
        Vec2 e2 = data.p0 - data.p2;
        Vec2 v1 = uv - data.p1;
        Vec2 v2 = uv - data.p2;
        const auto dot1 = glm::dot(v1, e1);
        const auto dot2 = glm::dot(v2, e2);
        const auto dot3 = glm::dot(e1, e1);
        const auto dot4 = glm::dot(e2, e2);
        const auto div1 = dot1 / dot3;
        const auto div2 = dot2 / dot4;
        const auto clamp1 = glm::clamp(div1, Scalar(0), Scalar(1));
        const auto clamp2 = glm::clamp(div2, Scalar(0), Scalar(1));
        Vec2 pq1 = v1 - e1 * clamp1;
        Vec2 pq2 = v2 - e2 * clamp2;
        Scalar s = glm::sign(e0.x * e2.y - e0.y * e2.x);
        Vec2 d0 = Vec2(glm::dot(pq0, pq0), s * (v0.x * e0.y - v0.y * e0.x));
        Vec2 d1 = Vec2(glm::dot(pq1, pq1), s * (v1.x * e1.y - v1.y * e1.x));
        Vec2 d2 = Vec2(glm::dot(pq2, pq2), s * (v2.x * e2.y - v2.y * e2.x));
        Vec2 d = min(min(d0, d1), d2);

        poly_sign = -glm::sign(d.y);
        distance_sq = d.x;
    } else {
        distance_sq = dot(pq0, pq0);
    }

    // return sqrt(distance_sq) * poly_sign;
    const auto sdf_val = sqrt(distance_sq) * poly_sign;

    const auto grad_sqrt_V = incoming_grad * poly_sign;

    // no grad for poly_sign
    const auto grad_distance_sq = stroke::grad::sqrt(distance_sq, grad_sqrt_V);
    Vec2 grad_uv = {};
    Vec2 grad_pq0 = {};
    if (data.is_polygon) {
        Vec2 e1 = data.p2 - data.p1;
        Vec2 e2 = data.p0 - data.p2;
        Vec2 v1 = uv - data.p1;
        Vec2 v2 = uv - data.p2;
        const auto dot1 = glm::dot(v1, e1);
        const auto dot2 = glm::dot(v2, e2);
        const auto one_over_dot1 = 1 / glm::dot(e1, e1);
        const auto one_over_dot2 = 1 / glm::dot(e2, e2);
        const auto div1 = dot1 * one_over_dot1;
        const auto div2 = dot2 * one_over_dot2;
        const auto clamp1 = glm::clamp(div1, Scalar(0), Scalar(1));
        const auto clamp2 = glm::clamp(div2, Scalar(0), Scalar(1));
        Vec2 pq1 = v1 - e1 * clamp1;
        Vec2 pq2 = v2 - e2 * clamp2;
        Scalar s = glm::sign(e0.x * e2.y - e0.y * e2.x);
        Vec2 d0 = Vec2(glm::dot(pq0, pq0), s * (v0.x * e0.y - v0.y * e0.x));
        Vec2 d1 = Vec2(glm::dot(pq1, pq1), s * (v1.x * e1.y - v1.y * e1.x));
        Vec2 d2 = Vec2(glm::dot(pq2, pq2), s * (v2.x * e2.y - v2.y * e2.x));
        Vec2 d = min(min(d0, d1), d2);

        // gradient computation
        // result = d.x;
        Scalar grad_d0_x = 0;
        Scalar grad_d1_x = 0;
        Scalar grad_d2_x = 0;
        // Scalar d = min(min(d0, d1), d2);
        if (d0.x <= d1.x && d0.x <= d2.x) {
            grad_d0_x = grad_distance_sq;
        } else if (d1.x < d0.x && d1.x <= d2.x) {
            grad_d1_x = grad_distance_sq;
        } else {
            assert(d2.x <= d0.x && d2.x <= d1.x);
            grad_d2_x = grad_distance_sq;
        }
        grad_pq0 = grad_self_dot(pq0, grad_d0_x);
        const auto grad_pq1 = grad_self_dot(pq1, grad_d1_x);
        const auto grad_pq2 = grad_self_dot(pq2, grad_d2_x);

        // Vec2 pq1 = v1 - e1 * clamp1;
        auto grad_v1 = grad_pq1;
        auto grad_e1 = -grad_pq1 * clamp1;
        auto grad_clamp1 = -glm::dot(e1, grad_pq1);

        // Vec2 pq2 = v2 - e2 * clamp2;
        auto grad_v2 = grad_pq2;
        auto grad_e2 = -grad_pq2 * clamp2;
        auto grad_clamp2 = -glm::dot(e2, grad_pq2);

        // const auto clamp1 = glm::clamp(div1, Scalar(0), Scalar(1));
        auto grad_div1 = stroke::grad::clamp(div1, Scalar(0), Scalar(1), grad_clamp1);

        // const auto clamp2 = glm::clamp(div2, Scalar(0), Scalar(1));
        auto grad_div2 = stroke::grad::clamp(div2, Scalar(0), Scalar(1), grad_clamp2);

        // const auto div1 = dot1 * one_over_dot1;
        const auto grad_dot1 = grad_div1 * one_over_dot1;

        // const auto div2 = dot2 * one_over_dot2;
        const auto grad_dot2 = grad_div2 * one_over_dot2;

        // const auto dot1 = glm::dot(v1, e1);
        stroke::grad::dot(v1, e1, grad_dot1).addTo(&grad_v1, &grad_e1);
        // const auto dot2 = glm::dot(v2, e2);
        stroke::grad::dot(v2, e2, grad_dot2).addTo(&grad_v2, &grad_e2);

        // continue with
        grad_uv += grad_v1 + grad_v2;

    } else {
        // result = dot(pq0, pq0);
        grad_pq0 += grad_self_dot(pq0, grad_distance_sq);
    }

    Vec2 grad_v0 = grad_pq0;
    const auto grad_clamp = -glm::dot(grad_pq0, e0);
    const auto grad_div0 = stroke::grad::clamp(div0, Scalar(0), Scalar(1), grad_clamp);
    const auto grad_dot0 = grad_div0 * one_over_dot0;
    stroke::grad::dot(v0, e0, grad_dot0).addTo(&grad_v0, stroke::grad::Ignore::Grad);
    grad_uv += grad_v0;

    return Vec3(sdf_val, grad_uv);
};

} // namespace alpine

TEST_CASE("alpine maps sdf")
{
    SECTION("intersect_with_ray_inv_C")
    {

        std::vector<alpine::GeomData> geomdata = {
            { { 0.0, 0.0 }, { 1.0, 1.0 }, { 0.0, 1.0 }, true }, // left bottom triangle ( one edge goes through center)
            { { 0.0, 0.0 }, { 0.5, 1.0 }, { 1.0, 0.5 }, true }, // triangle that encloses center
            { { 0.0, 0.0 }, { 0.2, 1.0 }, { 0.0, 1.0 }, true }, // triangle that is away from center

            { { 0.0, 0.0 }, { 0.0, 1.0 }, { 0.0, 0.0 }, false }, // horizontal line at uv border
            { { 0.0, 0.0 }, { 1.0, 0.0 }, { 0.0, 0.0 }, false }, // vertical line at uv border
            { { 0.0, 0.0 }, { 1.0, 1.0 }, { 0.0, 0.0 }, false }, // diagonal line through center

            { { 0.06, -0.04 }, { 0.7, 0.62 }, { 0.46, 0.78 }, true },
            { { 0.58, 1.1 }, { 0.9, 0.94 }, { -0.08, 0.38 }, true },
            { { 0.22, 0.54 }, { 0.7, 0 }, { 0.62, 0.82 }, true },
            { { 0.82, 0.8 }, { 0.52, -0.02 }, { 0.32, 1.12 }, true },
            { { 0.44, 0.22 }, { 0.82, 0.04 }, { 0.78, 0.38 }, true },
            { { 0.86, 0.88 }, { -0.02, 0.12 }, { 0.4, 0.8 }, true },
            { { 0.78, 1.16 }, { -0.06, 0.1 }, { 0.5, 0.38 }, true },
            { { 1, 0.24 }, { -0.06, 0.84 }, { 0.32, -0.04 }, true },
            { { 0.54, 1.02 }, { 0.94, 1.02 }, { 0.86, 1.12 }, true },
            { { 0.8, 0.6 }, { 0.28, 0.82 }, { 1, 0.98 }, true },
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0.92, -0.02 }, true },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0.22, 1.18 }, true },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0.64, 0.08 }, true },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 1.16, 0.22 }, true },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0.5, 1.04 }, true },
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0.14, 0.82 }, true },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0.96, 1.06 }, true },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0.66, 0.82 }, true },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0.86, 0.82 }, true },
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0, 0 }, false },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0, 0 }, false },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0, 0 }, false },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 0, 0 }, false },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0, 0 }, false },
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0, 0 }, false },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0, 0 }, false },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0, 0 }, false },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0, 0 }, false },
        };

        whack::random::HostGenerator<Scalar> rnd;

        for (const auto& data : geomdata) {
            for (int i = 0; i < 10; ++i) {
                const auto fun = [&](const whack::Tensor<Scalar, 1>& input) {
                    const auto uv = stroke::extract<Vec2>(input);
                    const auto d = alpine::sdf(data, uv);
                    const auto d2 = sdf_with_grad(data, uv, 1);
                    CHECK(Catch::Approx(d) == d2.x);
                    return stroke::pack_tensor<Scalar>(d);
                };

                const auto fun_grad = [&](const whack::Tensor<Scalar, 1>& input, const whack::Tensor<Scalar, 1>& grad_output) {
                    const auto uv = stroke::extract<Vec2>(input);
                    const auto grad_incoming = stroke::extract<Scalar>(grad_output);

                    const auto grad_outgoing = alpine::sdf_with_grad(data, uv, grad_incoming);

                    return stroke::pack_tensor<Scalar>(grad_outgoing.y, grad_outgoing.z);
                };

                const auto uv = rnd.uniform2();
                const auto test_data = stroke::pack_tensor<Scalar>(uv);
                stroke::check_gradient(fun, fun_grad, test_data, Scalar(0.000001));
            }
        }
    }
}
