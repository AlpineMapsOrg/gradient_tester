#include "hip/hip_runtime.h"
/****************************************************************************
 *  Copyright (C) 2023 Adam Celarek (github.com/adam-ce, github.com/cg-tuwien)
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy
 *  of this software and associated documentation files (the "Software"), to deal
 *  in the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 *  of the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in
 *  all copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *  THE SOFTWARE.
 ****************************************************************************/

#include <stroke/unittest/gradcheck.h>

#include <catch2/catch_test_macros.hpp>
#include <stroke/gaussian.h>
#include <stroke/grad/gaussian.h>
#include <stroke/linalg.h>
#include <stroke/unittest/random_entity.h>

using Scalar = double;
using Vec2 = glm::vec<2, Scalar>;
using Vec3 = glm::vec<3, Scalar>;

namespace alpine {
struct GeomData {
    Vec2 p0;
    Vec2 p1;
    Vec2 p2;

    bool is_polygon;

    bool line_cap0;
    bool line_cap1;

    // normally in style, but for this test putting it here should be good enough
    Scalar line_width;
    Vec2 dash_info;
    bool round_line_caps;
};

Scalar sdf(const GeomData& data, const Vec2& uv, Scalar line_width, Vec2 dash_info, bool round_line_caps)
{
    Vec2 e0 = data.p1 - data.p0;
    Vec2 v0 = uv - data.p0;
    Vec2 v1 = uv - data.p1;

    Scalar h = glm::clamp(glm::dot(v0, e0) / glm::dot(e0, e0), Scalar(0), Scalar(1));
    Vec2 pq0 = v0 - e0 * h;

    Scalar poly_sign = 1.0;
    Scalar result = 1.0;
    Scalar mask = 1.0;

    if (data.is_polygon) {
        Vec2 e1 = data.p2 - data.p1;
        Vec2 e2 = data.p0 - data.p2;
        Vec2 v2 = uv - data.p2;
        Vec2 pq1 = v1 - e1 * glm::clamp(glm::dot(v1, e1) / glm::dot(e1, e1), Scalar(0), Scalar(1));
        Vec2 pq2 = v2 - e2 * glm::clamp(glm::dot(v2, e2) / glm::dot(e2, e2), Scalar(0), Scalar(1));
        Scalar s = glm::sign(e0.x * e2.y - e0.y * e2.x);
        Vec2 d0 = Vec2(glm::dot(pq0, pq0), s * (v0.x * e0.y - v0.y * e0.x));
        Vec2 d1 = Vec2(glm::dot(pq1, pq1), s * (v1.x * e1.y - v1.y * e1.x));
        Vec2 d2 = Vec2(glm::dot(pq2, pq2), s * (v2.x * e2.y - v2.y * e2.x));
        Vec2 d = min(min(d0, d1), d2);

        poly_sign = -glm::sign(d.y);
        result = d.x;
    } else {
        Scalar line_length = glm::length(e0);

        Scalar amount_dash_gap_pairs = glm::ceil(line_length/dash_info.y);
        // + 0.01 -> small delta to remove artifacts if there shouldn't be any dashes
        Scalar dash_period = cos(M_PI*h*amount_dash_gap_pairs*Scalar(2))+cos((Scalar(1)-dash_info.x)*M_PI)+Scalar(0.01);
        // tanh is used as a differentiable step function -> all values above 0 are mapped to +1, all below to -1
        // multiplication by big value ensures a quick transition at 0 +/- small delta
        Scalar dashes = tanh(dash_period*Scalar(500000.0));

        Scalar line_endings = 1.0;
        if(!round_line_caps)
        {
            if(data.line_cap0)
                line_endings *= dot(normalize(e0), v0);
            if(data.line_cap1)
                line_endings *= dot(normalize(-e0), v1);
        }
        line_endings = (tanh(line_endings*Scalar(500000.0))+Scalar(1)) / Scalar(2);

        mask = line_endings*dashes;

        result = glm::dot(pq0, pq0);
    }

    return (sqrt(result) * poly_sign) - (line_width * mask);
};

Vec3 sdf_with_grad(const GeomData& data, const Vec2& uv, Scalar incoming_grad)
{
    const Vec2 e0 = data.p1 - data.p0;
    const Vec2 v0 = uv - data.p0;
    const auto dot0 = glm::dot(v0, e0);
    const auto one_over_dot0 = 1 / glm::dot(e0, e0);
    const auto div0 = dot0 * one_over_dot0;
    const Vec2 pq0 = v0 - e0 * glm::clamp(div0, Scalar(0), Scalar(1));
    const auto dot_pq0_pq0 = glm::dot(pq0, pq0);

    Scalar poly_sign = 1.0;
    Scalar distance_sq = 1.0;

    Vec2 grad_uv = {};
    Vec2 grad_pq0 = {};
    if (data.is_polygon) {
        const Vec2 e1 = data.p2 - data.p1;
        const Vec2 e2 = data.p0 - data.p2;
        const Vec2 v1 = uv - data.p1;
        const Vec2 v2 = uv - data.p2;
        const auto dot1 = glm::dot(v1, e1);
        const auto dot2 = glm::dot(v2, e2);
        const auto one_over_dot1 = 1 / glm::dot(e1, e1);
        const auto one_over_dot2 = 1 / glm::dot(e2, e2);
        const auto div1 = dot1 * one_over_dot1;
        const auto div2 = dot2 * one_over_dot2;
        const auto clamp1 = glm::clamp(div1, Scalar(0), Scalar(1));
        const auto clamp2 = glm::clamp(div2, Scalar(0), Scalar(1));
        const Vec2 pq1 = v1 - e1 * clamp1;
        const Vec2 pq2 = v2 - e2 * clamp2;
        const Scalar s = glm::sign(e0.x * e2.y - e0.y * e2.x);
        const Vec2 d0 = Vec2(dot_pq0_pq0, s * (v0.x * e0.y - v0.y * e0.x));
        const Vec2 d1 = Vec2(glm::dot(pq1, pq1), s * (v1.x * e1.y - v1.y * e1.x));
        const Vec2 d2 = Vec2(glm::dot(pq2, pq2), s * (v2.x * e2.y - v2.y * e2.x));
        const Vec2 d = min(min(d0, d1), d2);

        poly_sign = -glm::sign(d.y);
        distance_sq = d.x;

        // gradient computation
        Scalar grad_d0_x = 0;
        Scalar grad_d1_x = 0;
        Scalar grad_d2_x = 0;

        if (d0.x <= d1.x && d0.x <= d2.x) {
            grad_d0_x = incoming_grad;
        } else if (d1.x < d0.x && d1.x <= d2.x) {
            grad_d1_x = incoming_grad;
        } else {
            assert(d2.x <= d0.x && d2.x <= d1.x);
            grad_d2_x = incoming_grad;
        }
        grad_pq0 = Scalar(2) * pq0 * grad_d0_x;
        const auto grad_pq1 = Scalar(2) * pq1 * grad_d1_x;
        const auto grad_pq2 = Scalar(2) * pq2 * grad_d2_x;

        auto grad_v1 = grad_pq1;
        auto grad_e1 = -grad_pq1 * clamp1;
        const auto grad_clamp1 = -glm::dot(e1, grad_pq1);

        auto grad_v2 = grad_pq2;
        auto grad_e2 = -grad_pq2 * clamp2;
        const auto grad_clamp2 = -glm::dot(e2, grad_pq2);

        const auto grad_div1 = stroke::grad::clamp(div1, Scalar(0), Scalar(1), grad_clamp1);

        const auto grad_div2 = stroke::grad::clamp(div2, Scalar(0), Scalar(1), grad_clamp2);

        const auto grad_dot1 = grad_div1 * one_over_dot1;

        const auto grad_dot2 = grad_div2 * one_over_dot2;

        grad_v1 += e1 * grad_dot1;
        grad_e1 += v1 * grad_dot1;

        grad_v2 += e2 * grad_dot2;
        grad_e2 += v2 * grad_dot2;

        grad_uv += grad_v1 + grad_v2;

    } else {
        grad_pq0 += Scalar(2) * pq0 * incoming_grad;
        distance_sq = dot_pq0_pq0;
    }

    Vec2 grad_v0 = grad_pq0;
    const auto grad_clamp = -glm::dot(grad_pq0, e0);
    const auto grad_div0 = stroke::grad::clamp(div0, Scalar(0), Scalar(1), grad_clamp);
    const auto grad_dot0 = grad_div0 * one_over_dot0;
    grad_v0 += e0 * grad_dot0;
    grad_uv += grad_v0;
    const auto sdf_val = sqrt(distance_sq) * poly_sign;

    return Vec3(sdf_val, grad_uv / (2 * sdf_val));
};

} // namespace alpine

TEST_CASE("alpine maps sdf")
{
    SECTION("intersect_with_ray_inv_C")
    {

        std::vector<alpine::GeomData> geomdata = {
            { { 0.0, 0.0 }, { 1.0, 1.0 }, { 0.0, 1.0 }, true, false, false, 0.0, {1.0, 1.0}, true }, // left bottom triangle ( one edge goes through center)
            { { 0.0, 0.0 }, { 0.5, 1.0 }, { 1.0, 0.5 }, true, false, false, 0.0, {1.0, 1.0}, true }, // triangle that encloses center
            { { 0.0, 0.0 }, { 0.2, 1.0 }, { 0.0, 1.0 }, true, false, false, 0.0, {1.0, 1.0}, true }, // triangle that is away from center

            { { 0.0, 0.0 }, { 0.0, 1.0 }, { 0.0, 0.0 }, false, false, false }, // horizontal line at uv border
            { { 0.0, 0.0 }, { 1.0, 0.0 }, { 0.0, 0.0 }, false, false, false }, // vertical line at uv border
            { { 0.0, 0.0 }, { 1.0, 1.0 }, { 0.0, 0.0 }, false, false, false }, // diagonal line through center

            { { 0.0, 0.0 }, { 0.0, 1.0 }, { 0.0, 0.0 }, false, true, true}, // horizontal line at uv border
            { { 0.0, 0.0 }, { 1.0, 0.0 }, { 0.0, 0.0 }, false, true, true }, // vertical line at uv border
            { { 0.0, 0.0 }, { 1.0, 1.0 }, { 0.0, 0.0 }, false, true, true }, // diagonal line through center

            { { 0.0, 0.0 }, { 0.0, 1.0 }, { 0.0, 0.0 }, false, true, false}, // horizontal line at uv border
            { { 0.0, 0.0 }, { 1.0, 0.0 }, { 0.0, 0.0 }, false, false, true }, // vertical line at uv border
            { { 0.0, 0.0 }, { 1.0, 1.0 }, { 0.0, 0.0 }, false, true, false }, // diagonal line through center

            { { 0.06, -0.04 }, { 0.7, 0.62 }, { 0.46, 0.78 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.58, 1.1 }, { 0.9, 0.94 }, { -0.08, 0.38 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.22, 0.54 }, { 0.7, 0 }, { 0.62, 0.82 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.82, 0.8 }, { 0.52, -0.02 }, { 0.32, 1.12 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.44, 0.22 }, { 0.82, 0.04 }, { 0.78, 0.38 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.86, 0.88 }, { -0.02, 0.12 }, { 0.4, 0.8 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.78, 1.16 }, { -0.06, 0.1 }, { 0.5, 0.38 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 1, 0.24 }, { -0.06, 0.84 }, { 0.32, -0.04 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.54, 1.02 }, { 0.94, 1.02 }, { 0.86, 1.12 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.8, 0.6 }, { 0.28, 0.82 }, { 1, 0.98 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0.92, -0.02 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0.22, 1.18 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0.64, 0.08 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 1.16, 0.22 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0.5, 1.04 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0.14, 0.82 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0.96, 1.06 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0.66, 0.82 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0.86, 0.82 }, true, false, false, 0.0, {1.0, 1.0}, true },
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, true },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, true },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, true },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, true },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, true },
            // line endings -> but not at line segment end
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, false },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, false },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, false },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0, 0 }, false, false, false, 0.05, {1.0, 1.0}, false },

            // line endings
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0, 0 }, false, true, true, 0.05, {1.0, 1.0}, false },

            // dashes
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0, 0 }, false, false, false, 0.05, {0.5, 0.05}, true },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0, 0 }, false, false, false, 0.05, {0.7, 0.05}, true },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0, 0 }, false, false, false, 0.05, {0.5, 0.5}, true },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 0, 0 }, false, false, false, 0.05, {0.5, 0.002}, true },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0, 0 }, false, false, false, 0.05, {0.5, 0.05}, true },
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0, 0 }, false, false, false, 0.05, {0.5, 0.0005}, true },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0, 0 }, false, false, false, 0.05, {0.2, 0.05}, true },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0, 0 }, false, false, false, 0.05, {0.3, 0.05}, true },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0, 0 }, false, false, false, 0.05, {0.4, 0.005}, true },

            // dashes + line endings
            { { 0, 0.56 }, { 0.18, 0.54 }, { 0, 0 }, false, true, true, 0.05, {0.5, 0.05}, false },
            { { 0.76, 1.02 }, { 0.14, 0.88 }, { 0, 0 }, false, true, true, 0.05, {0.7, 0.05}, false },
            { { 1.16, 0.66 }, { 0.42, 1.1 }, { 0, 0 }, false, true, true, 0.05, {0.5, 0.5}, false },
            { { 0.44, 0.44 }, { 0.68, 0.56 }, { 0, 0 }, false, true, true, 0.05, {0.5, 0.002}, false },
            { { -0.1, 0.26 }, { 1, 0.18 }, { 0, 0 }, false, true, true, 0.05, {0.5, 0.05}, false },
            { { 0.24, 1.16 }, { 0.7, 0.56 }, { 0, 0 }, false, true, true, 0.05, {0.5, 0.0005}, false },
            { { 0, 0.28 }, { 1.18, 0.58 }, { 0, 0 }, false, true, true, 0.05, {0.2, 0.05}, false },
            { { 0.7, 0.78 }, { 0.54, 0.48 }, { 0, 0 }, false, true, true, 0.05, {0.3, 0.05}, false },
            { { 0.22, -0.1 }, { 0.78, 1 }, { 0, 0 }, false, true, true, 0.05, {0.4, 0.005}, false },
        };

        whack::random::HostGenerator<Scalar> rnd;

        for (const auto& data : geomdata) {
            for (int i = 0; i < 10; ++i) {
                const auto fun = [&](const whack::Tensor<Scalar, 1>& input) {
                    const auto uv = stroke::extract<Vec2>(input);
                    const auto d = alpine::sdf(data, uv, data.line_width, data.dash_info, data.round_line_caps);
                    const auto d2 = sdf_with_grad(data, uv, 1);
                    CHECK(Catch::Approx(d) == d2.x);
                    return stroke::pack_tensor<Scalar>(d);
                };

                const auto fun_grad = [&](const whack::Tensor<Scalar, 1>& input, const whack::Tensor<Scalar, 1>& grad_output) {
                    const auto uv = stroke::extract<Vec2>(input);
                    const auto grad_incoming = stroke::extract<Scalar>(grad_output);

                    const auto grad_outgoing = alpine::sdf_with_grad(data, uv, grad_incoming);

                    return stroke::pack_tensor<Scalar>(grad_outgoing.y, grad_outgoing.z);
                };

                const auto uv = rnd.uniform2();
                const auto test_data = stroke::pack_tensor<Scalar>(uv);
                stroke::check_gradient(fun, fun_grad, test_data, Scalar(0.000001));
            }
        }
    }
}
